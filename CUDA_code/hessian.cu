#include "hip/hip_runtime.h"
#include <stdint.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define CHANNEL_NUM 3

const int N = 16; 
const int blocksize = 16; 


__global__ void convert_to_grayscale(uint8_t *a, uint8_t *b) 
{

    int blockId = blockIdx.x + blockIdx.y * gridDim.x; 
    int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    int pos = threadId;
    int g_pos = pos * 3;

    // RGB to monochrome formula: (0.2125 * red) + (0.7154 * green) + (0.0721 * blue)
    b[pos] = a[g_pos+1] * 0.2125; //RED
    b[pos] += a[g_pos+2] * 0.7154; //GREEN
    b[pos] += a[g_pos+3] * 0.0721; //BLUE


}
 


__global__ void hessian(uint8_t const  __restrict__ *b, uint8_t *h, int8_t const  __restrict__ *f1){


    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    int sums = 0;  // will be private type
    //TODO: width height 4er piksel daha dar verilmeli
    int width = f1[0];  // will be private type
    int heigth = f1[1];  // will be private type

    int pos = threadId + 4 * width + 4; // will be private type
    int f_pos = 2; // will be private type
    int pos_buffer = pos; // will be private type

    for (int order =0; order < 4 ; order++ ){

        sums =0;
    
        for (int i =0; i< 9; i++){
            for(int j =0; j< 9; j++){
    
               // forward to the next pixel
               sums += b[pos_buffer + j] * f1[f_pos++];
    
    
            }
    
            // go to the bottom line of the photo 
            pos_buffer += width;
    
        }
    
        // normalize filter result by filter coefficients
        h[pos*4 + order] = sums / f1[f_pos++];


    }
}


int main() {


    int width, height, bpp;

    float elapsed=0;
    hipEvent_t start, stop;

    uint8_t* rgb_image = stbi_load("1.jpg", &width, &height, &bpp, 3);

    int8_t f1[324] ={ 0,0,1,1,1,1,1,0,0,
                     0,0,1,1,1,1,1,0,0,
                     0,0,1,1,1,1,1,0,0,
                     0,0,-2,-2,-2,-2,-2,0,0,
                     0,0,-2,-2,-2,-2,-2,0,0,
                     0,0,-2,-2,-2,-2,-2,0,0,
                     0,0,1,1,1,1,1,0,0,
                     0,0,1,1,1,1,1,0,0,
                     0,0,1,1,1,1,1,0,0,
                     0,0,0,0,0,0,0,0,0,
                     0,0,0,0,0,0,0,0,0,
                     1,1,1,-2,-2,-2,1,1,1,
                     1,1,1,-2,-2,-2,1,1,1,
                     1,1,1,-2,-2,-2,1,1,1,
                     1,1,1,-2,-2,-2,1,1,1,
                     1,1,1,-2,-2,-2,1,1,1,
                     0,0,0,0,0,0,0,0,0,
                     0,0,0,0,0,0,0,0,0,
                     0,0,0,0,0,0,0,0,0,
                     0,1,1,1,0,-1,-1,-1,0,
                     0,1,1,1,0,-1,-1,-1,0,
                     0,1,1,1,0,-1,-1,-1,0,
                     0,0,0,0,0,0,0,0,0,
                     0,-1,-1,-1,0,1,1,1,0,
                     0,-1,-1,-1,0,1,1,1,0,
                     0,-1,-1,-1,0,1,1,1,0,
                     0,0,0,0,0,0,0,0,0,
                     0,0,0,0,0,0,0,0,0,
                     0,-1,-1,-1,0,1,1,1,0,
                     0,-1,-1,-1,0,1,1,1,0,
                     0,-1,-1,-1,0,1,1,1,0,
                     0,0,0,0,0,0,0,0,0,
                     0,1,1,1,0,-1,-1,-1,0,
                     0,1,1,1,0,-1,-1,-1,0,
                     0,1,1,1,0,-1,-1,-1,0,
                     0,0,0,0,0,0,0,0,0,
                    };


    int im_size = width * height;

    uint8_t* return_rgb_image = (uint8_t*)malloc(im_size * 4);


 
    uint8_t *c_image;
    uint8_t *return_image;
    uint8_t *hessian_image;
    int8_t *g_f1;
 
 
    hipMalloc( (void**)&c_image, im_size * CHANNEL_NUM ); 
    hipMalloc( (void**)&return_image, im_size); 
    hipMalloc( (void**)&hessian_image, im_size*4); 
    hipMalloc( (void**)&g_f1, 324); 

    dim3 dimBlock( blocksize, blocksize );
    dim3 dimGrid( (width-4)/blocksize, (height-4)/blocksize );

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    for (int i =0 ; i< 60; i++){



        hipMemcpy( c_image, rgb_image, im_size * CHANNEL_NUM, hipMemcpyHostToDevice ); 
        hipMemcpy( g_f1, f1, 81, hipMemcpyHostToDevice ); 

        convert_to_grayscale<<<dimGrid, dimBlock>>>(c_image, return_image);
        hessian<<<dimGrid, dimBlock>>>(return_image, hessian_image, g_f1);


        hipMemcpy( return_rgb_image, hessian_image, im_size*4, hipMemcpyDeviceToHost ); 
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize (stop) ;

    hipEventElapsedTime(&elapsed, start, stop) ;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("time: %.2f ms\n", elapsed);

    hipFree( c_image );
    hipFree( return_image );
    hipFree( hessian_image );
    
    stbi_write_png("image.png", width, height, 1, return_rgb_image, width);
    stbi_image_free(rgb_image);
    stbi_image_free(return_rgb_image);

    return EXIT_SUCCESS;
}
